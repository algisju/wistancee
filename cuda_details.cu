#include <memory>
#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
// Main Program
using namespace std;
int main(void)
{
int device_Count = 0;
hipGetDeviceCount(&device_Count);
if (device_Count == 0)
	{
	printf("There are no available device(s) that support CUDA\n");
	}
else
	{
	printf("Detected %d CUDA Capable device(s)\n", device_Count);
	}
int device=0;int driver_Version=0;int runtime_Version=0;
hipDeviceProp_t device_Property;
hipGetDeviceProperties(&device_Property, device);
printf("\nDevice %d: \"%s\"\n", device, device_Property.name);
hipDriverGetVersion(&driver_Version);
hipRuntimeGetVersion(&runtime_Version);
printf(" CUDA Driver Version / Runtime Version %d.%d / %d.%d\n", driver_Version / 1000, (driver_Version% 100) / 10, runtime_Version / 1000, (runtime_Version % 100) / 10);
printf(" Total amount of global memory: %.0f MBytes (%llu bytes)\n",(float)device_Property.totalGlobalMem / 1048576.0f, (unsigned long long) device_Property.totalGlobalMem);
printf(" (%2d) Multiprocessors", device_Property.multiProcessorCount );
printf(" GPU Max Clock rate: %.0f MHz (%0.2f GHz)\n", device_Property.clockRate * 1e-3f,device_Property.clockRate * 1e-6f);
printf(" Maximum number of threads per multiprocessor: %d\n",device_Property.maxThreadsPerMultiProcessor);
printf(" Maximum number of threads per block: %d\n", device_Property.maxThreadsPerBlock);
printf(" Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
	device_Property.maxThreadsDim[0],
	device_Property.maxThreadsDim[1],
	device_Property.maxThreadsDim[2]);
printf(" Max dimension size of a grid size (x,y,z): (%d, %d, %d)\n",
	device_Property.maxGridSize[0],
	device_Property.maxGridSize[1],
	device_Property.maxGridSize[2]);
printf( " Total amount of global memory: %.0f MBytes (%llu bytes)\n",
	(float)device_Property.totalGlobalMem / 1048576.0f, 
	(unsigned long long)device_Property.totalGlobalMem);
printf(" Memory Clock rate: %.0f Mhz\n", device_Property.memoryClockRate * 1e-3f);
printf(" Memory Bus Width: %d-bit\n", device_Property.memoryBusWidth);
if (device_Property.l2CacheSize)
	{
	printf(" L2 Cache Size: %d bytes\n", device_Property.l2CacheSize);
	}	
printf(" Total amount of constant memory: %lu bytes\n", device_Property.totalConstMem);
printf(" Total amount of shared memory per block: %lu bytes\n", device_Property.sharedMemPerBlock);
printf(" Total number of registers available per block: %d\n", device_Property.regsPerBlock);
}
