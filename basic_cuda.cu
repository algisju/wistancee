#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
__global__ void gpuAdd(int d_a, int d_b, int *d_c)
{
	//execute in gpu
	*d_c = d_a + d_b;
}

int main(void)
{
	int h_c;
	int *d_c;
	hipMalloc((void**)&d_c, sizeof(int));
	gpuAdd << <1, 1 >> > (8, 1, d_c);
	hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("Yayaya 8 + 3 = %d\n", h_c);
	hipFree(d_c);
	return 0;
}
