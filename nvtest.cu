#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE	4096

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	int i = threadIdx.x;
	if (i < n)
		c[i] = a[i] + b[i];
}

int main()
{

	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	}


	int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	a = (int *)malloc(SIZE*sizeof(int));
	b = (int *)malloc(SIZE*sizeof(int));
	c = (int *)malloc(SIZE*sizeof(int));

	hipMalloc( &d_a, SIZE*sizeof(int));
	hipMalloc( &d_b, SIZE*sizeof(int));
	hipMalloc( &d_c, SIZE*sizeof(int));

	for( int i = 0; i < SIZE; ++i )
	{
		a[i] = i;
		b[i] = i;
		c[i] = i;
	}

	hipMemcpy( d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_c, c, SIZE*sizeof(int), hipMemcpyHostToDevice );

	VectorAdd<<< 1, SIZE >>>(d_a, d_b, d_c, SIZE);
	
	hipMemcpy( c, d_c, SIZE*sizeof(int), hipMemcpyDeviceToHost );

	for( int i = 0; i < 10; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
